#include "hip/hip_runtime.h"
#include <Tools/CUDA_Reallocate.cuh>
#include <Tools/CUDA_Configuration.cuh>
#include <CUDA/CUDA_Neural_Network.cuh>

__host__ __device__ CUDA_Device_Information_Array::CUDA_Device_Information_Array(void) { }

__global__ void kernel__Class_Device_Information_Array__Push_Back(int const index_device_received,
                                                                                                       struct hipDeviceProp_t *const ptr_struct_cudaDeviceProp_received,
                                                                                                       class CUDA_Device_Information_Array *const ptr_Class_Device_Information_Array_received)
{ ptr_Class_Device_Information_Array_received->Push_Back(index_device_received, ptr_struct_cudaDeviceProp_received); }

__host__ bool CUDA_Device_Information_Array::Push_Back(int const index_device_received)
{
    struct hipDeviceProp_t tmp_struct_cudaDeviceProp,
                                            *tmp_ptr_device_struct_cudaDeviceProp(NULL);

    CUDA__Safe_Call(hipGetDeviceProperties(&tmp_struct_cudaDeviceProp, index_device_received));

    CUDA__Safe_Call(hipMalloc((void**)&tmp_ptr_device_struct_cudaDeviceProp, sizeof(struct hipDeviceProp_t)));

    CUDA__Safe_Call(hipMemcpy(tmp_ptr_device_struct_cudaDeviceProp,
                                                            &tmp_struct_cudaDeviceProp,
                                                            sizeof(struct hipDeviceProp_t),
                                                            hipMemcpyKind::hipMemcpyHostToDevice));

    kernel__Class_Device_Information_Array__Push_Back <<< 1u, 1u >>> (index_device_received,
                                                                                                             tmp_ptr_device_struct_cudaDeviceProp,
                                                                                                             this);
        
#if defined(COMPILE_DEBUG)
    CUDA__Check_Error();
#endif

    CUDA__Safe_Call(hipFree(tmp_ptr_device_struct_cudaDeviceProp));

    return(true);
}
    
__host__ __device__ bool CUDA_Device_Information_Array::Push_Back(int const index_device_received, struct hipDeviceProp_t *const ptr_struct_cudaDeviceProp_received)
{
    if(ptr_struct_cudaDeviceProp_received == nullptr) { return(false); }

#if defined(__CUDA_ARCH__) == false
    kernel__Class_Device_Information_Array__Push_Back <<< 1u, 1u >>> (index_device_received,
                                                                                                             ptr_struct_cudaDeviceProp_received,
                                                                                                             this);
        
#if defined(COMPILE_DEBUG)
    CUDA__Check_Error();
#endif
#else
    if(CUDA__Required_Compatibility_Device(*ptr_struct_cudaDeviceProp_received))
    {
        for(size_t i(0); i != this->_number_cuda_devices; ++i)
        {
            if(this->_ptr_array_Class_Device_Information[i].Get__ID() == index_device_received)
            { return(true); }
        }

        if(this->_ptr_array_Class_Device_Information == nullptr)
        {
            this->_ptr_Class_Device_Information_sum = new class CUDA_Device_Information;
            this->_ptr_Class_Device_Information_higher = new class CUDA_Device_Information;
            this->_ptr_Class_Device_Information_lower = new class CUDA_Device_Information;
            this->_ptr_array_Class_Device_Information = new class CUDA_Device_Information[1u];
        }
        else
        {
            class CUDA_Device_Information *tmp_ptr_array_Class_Device_Information(Memory::reallocate_objects_cpp<class CUDA_Device_Information>(this->_ptr_array_Class_Device_Information,
                                                                                                                                                                                                                           this->_number_cuda_devices + 1u,
                                                                                                                                                                                                                           this->_number_cuda_devices));

            if(tmp_ptr_array_Class_Device_Information == nullptr)
            {
                PRINT_FORMAT("%s: ERROR: 'tmp_ptr_array_Class_Device_Information' is a nullptr." NEW_LINE, __FUNCTION__);

                return(false);
            }

            this->_ptr_array_Class_Device_Information = tmp_ptr_array_Class_Device_Information;
        }

        if(this->_ptr_array_Class_Device_Information[this->_number_cuda_devices].Initialize(index_device_received, ptr_struct_cudaDeviceProp_received))
        { this->Update(ptr_struct_cudaDeviceProp_received); }

        this->_selected_cuda_device = this->_number_cuda_devices;

        ++this->_number_cuda_devices;
    }
#endif

    return(true);
}

__global__ void kernel__Class_Device_Information_Array__Refresh(struct hipDeviceProp_t *const ptr_struct_cudaDeviceProp_received, class CUDA_Device_Information_Array *const ptr_Class_Device_Information_Array_received)
{ ptr_Class_Device_Information_Array_received->Update(ptr_struct_cudaDeviceProp_received); }

__host__ __device__ bool CUDA_Device_Information_Array::Update(struct hipDeviceProp_t *const ptr_struct_cudaDeviceProp_received)
{
    if(ptr_struct_cudaDeviceProp_received == nullptr) { return(false); }

#if defined(__CUDA_ARCH__) == false
    kernel__Class_Device_Information_Array__Refresh <<< 1u, 1u >>> (ptr_struct_cudaDeviceProp_received, this);
        
#if defined(COMPILE_DEBUG)
    CUDA__Check_Error();
#endif
#else
    PRINT_FORMAT("%s: [FUNCTION DEPRECATED] TODO: Fix \"Update\" algorithm." NEW_LINE, __FUNCTION__);

    // Sum += ptr_struct_cudaDeviceProp_received

    // Higher > ptr_struct_cudaDeviceProp_received

    // Lower < ptr_struct_cudaDeviceProp_received
#endif

    return(true);
}

__host__ __device__ bool CUDA_Device_Information_Array::Deallocate(void)
{
    SAFE_DELETE(this->_ptr_Class_Device_Information_sum);
    SAFE_DELETE(this->_ptr_Class_Device_Information_higher);
    SAFE_DELETE(this->_ptr_Class_Device_Information_lower);
    SAFE_DELETE_ARRAY(this->_ptr_array_Class_Device_Information);

    return(true);
}

__host__ __device__ bool CUDA_Device_Information_Array::Select_CUDA_Device(int const index_received)
{
    if(Get__Number_CUDA_Devices() > index_received)
    {
        this->_selected_cuda_device = index_received;

        return(true);
    }
    else
    {
        PRINT_FORMAT("%s: ERROR: Index overflow." NEW_LINE, __FUNCTION__);
            
        return(false);
    }
}

__host__ __device__ size_t CUDA_Device_Information_Array::Get__Number_CUDA_Devices(void) const { return(this->_number_cuda_devices); }

__host__ __device__ int CUDA_Device_Information_Array::Get__Selected_CUDA_Device(void) const { return(this->_selected_cuda_device); }

__host__ __device__ class CUDA_Device_Information* CUDA_Device_Information_Array::Get__CUDA_Device(void) const
{
    if(static_cast<int>(this->Get__Number_CUDA_Devices()) > this->_selected_cuda_device && this->_selected_cuda_device >= 0) { return(&this->_ptr_array_Class_Device_Information[this->_selected_cuda_device]); }
    else { return(nullptr); }
}

__host__ __device__ class CUDA_Device_Information* CUDA_Device_Information_Array::Get__CUDA_Device(size_t const index_received) const
{
    if(this->Get__Number_CUDA_Devices() > index_received) { return(&this->_ptr_array_Class_Device_Information[index_received]); }
    else { return(nullptr); }
}

__host__ __device__ CUDA_Device_Information_Array::~CUDA_Device_Information_Array(void)
{ this->Deallocate(); }

#include "hip/hip_runtime.h"
#include <Tools/CUDA_Configuration.cuh>
#include <CUDA/CUDA_Neural_Network.cuh>

template<typename T>
__global__ void kernel__Update_Parameter__Adam(T const beta1_received,
                                                                        T const beta2_received,
                                                                        T const epsilon_received,
                                                                        T const learning_rate_at_time_t_received,
                                                                        T *const ptr_array_partial_derivative_received,
                                                                        T *const ptr_array_parameters_received,
                                                                        T *const ptr_array_previous_biased_first_moment_received,
                                                                        T *const ptr_array_previous_biased_second_moment_received)
{
    size_t const tmp_thread_global_index(blockIdx.x * blockDim.x + threadIdx.x);
    
    T tmp_partial_derivative(ptr_array_partial_derivative_received[tmp_thread_global_index]),
       tmp_biased_first_moment,
       tmp_biased_second_moment;

    ptr_array_previous_biased_first_moment_received[tmp_thread_global_index] = tmp_biased_first_moment = beta1_received * ptr_array_previous_biased_first_moment_received[tmp_thread_global_index] + (T(1) - beta1_received) * tmp_partial_derivative;
    ptr_array_previous_biased_second_moment_received[tmp_thread_global_index] = tmp_biased_second_moment = beta2_received * ptr_array_previous_biased_second_moment_received[tmp_thread_global_index] + (T(1) - beta2_received) * tmp_partial_derivative * tmp_partial_derivative;

    ptr_array_parameters_received[tmp_thread_global_index] -= learning_rate_at_time_t_received * tmp_biased_first_moment / (sqrt(tmp_biased_second_moment) + epsilon_received);

    ptr_array_partial_derivative_received[tmp_thread_global_index] = T(0);
}

template<typename T>
__global__ void kernel__Update_Parameter__Adam(size_t const size_received,
                                                                        T const beta1_received,
                                                                        T const beta2_received,
                                                                        T const epsilon_received,
                                                                        T const learning_rate_at_time_t_received,
                                                                        T *const ptr_array_partial_derivative_received,
                                                                        T *const ptr_array_parameters_received,
                                                                        T *const ptr_array_previous_biased_first_moment_received,
                                                                        T *const ptr_array_previous_biased_second_moment_received)
{
    size_t const tmp_thread_global_index(blockIdx.x * blockDim.x + threadIdx.x);

    T tmp_partial_derivative,
       tmp_biased_first_moment,
       tmp_biased_second_moment;

    if(tmp_thread_global_index < size_received)
    {
        tmp_partial_derivative = ptr_array_partial_derivative_received[tmp_thread_global_index];

        ptr_array_previous_biased_first_moment_received[tmp_thread_global_index] = tmp_biased_first_moment = beta1_received * ptr_array_previous_biased_first_moment_received[tmp_thread_global_index] + (T(1) - beta1_received) * tmp_partial_derivative;
        ptr_array_previous_biased_second_moment_received[tmp_thread_global_index] = tmp_biased_second_moment = beta2_received * ptr_array_previous_biased_second_moment_received[tmp_thread_global_index] + (T(1) - beta2_received) * tmp_partial_derivative * tmp_partial_derivative;

        ptr_array_parameters_received[tmp_thread_global_index] -= learning_rate_at_time_t_received * tmp_biased_first_moment / (sqrt(tmp_biased_second_moment) + epsilon_received);

        ptr_array_partial_derivative_received[tmp_thread_global_index] = T(0);
    }
}

template<typename T>
__global__ void kernel_while__Update_Parameter__Adam(size_t const size_received,
                                                                                T const beta1_received,
                                                                                T const beta2_received,
                                                                                T const epsilon_received,
                                                                                T const learning_rate_at_time_t_received,
                                                                                T *const ptr_array_partial_derivative_received,
                                                                                T *const ptr_array_parameters_received,
                                                                                T *const ptr_array_previous_biased_first_moment_received,
                                                                                T *const ptr_array_previous_biased_second_moment_received)
{
    size_t tmp_thread_global_index(blockIdx.x * blockDim.x + threadIdx.x);
    
    T tmp_partial_derivative,
       tmp_biased_first_moment,
       tmp_biased_second_moment;

    do
    {
        tmp_partial_derivative = ptr_array_partial_derivative_received[tmp_thread_global_index];

        ptr_array_previous_biased_first_moment_received[tmp_thread_global_index] = tmp_biased_first_moment = beta1_received * ptr_array_previous_biased_first_moment_received[tmp_thread_global_index] + (T(1) - beta1_received) * tmp_partial_derivative;
        ptr_array_previous_biased_second_moment_received[tmp_thread_global_index] = tmp_biased_second_moment = beta2_received * ptr_array_previous_biased_second_moment_received[tmp_thread_global_index] + (T(1) - beta2_received) * tmp_partial_derivative * tmp_partial_derivative;

        ptr_array_parameters_received[tmp_thread_global_index] -= learning_rate_at_time_t_received * tmp_biased_first_moment / (sqrt(tmp_biased_second_moment) + epsilon_received);

        ptr_array_partial_derivative_received[tmp_thread_global_index] = T(0);

        tmp_thread_global_index += gridDim.x * blockDim.x;
    } while(tmp_thread_global_index < size_received);
}

template<typename T>
__global__ void kernel__Update_Parameter__Adam(T const weight_decay_received,
                                                                        T const beta1_received,
                                                                        T const beta2_received,
                                                                        T const epsilon_received,
                                                                        T const learning_rate_at_time_t_received,
                                                                        T const *const ptr_array_connections_mask_rergularization_received,
                                                                        T *const ptr_array_partial_derivative_received,
                                                                        T *const ptr_array_parameters_received,
                                                                        T *const ptr_array_previous_biased_first_moment_received,
                                                                        T *const ptr_array_previous_biased_second_moment_received)
{
    size_t const tmp_thread_global_index(blockIdx.x * blockDim.x + threadIdx.x);
    
    T tmp_partial_derivative(ptr_array_partial_derivative_received[tmp_thread_global_index]),
       tmp_biased_first_moment,
       tmp_biased_second_moment;

    ptr_array_previous_biased_first_moment_received[tmp_thread_global_index] = tmp_biased_first_moment = beta1_received * ptr_array_previous_biased_first_moment_received[tmp_thread_global_index] + (T(1) - beta1_received) * tmp_partial_derivative;
    ptr_array_previous_biased_second_moment_received[tmp_thread_global_index] = tmp_biased_second_moment = beta2_received * ptr_array_previous_biased_second_moment_received[tmp_thread_global_index] + (T(1) - beta2_received) * tmp_partial_derivative * tmp_partial_derivative;

    ptr_array_parameters_received[tmp_thread_global_index] -= learning_rate_at_time_t_received * tmp_biased_first_moment / (sqrt(tmp_biased_second_moment) + epsilon_received) + ptr_array_connections_mask_rergularization_received[tmp_thread_global_index] * weight_decay_received * ptr_array_parameters_received[tmp_thread_global_index];

    ptr_array_partial_derivative_received[tmp_thread_global_index] = T(0);
}

template<typename T>
__global__ void kernel__Update_Parameter__Adam(size_t const size_received,
                                                                        T const weight_decay_received,
                                                                        T const beta1_received,
                                                                        T const beta2_received,
                                                                        T const epsilon_received,
                                                                        T const learning_rate_at_time_t_received,
                                                                        T const *const ptr_array_connections_mask_rergularization_received,
                                                                        T *const ptr_array_partial_derivative_received,
                                                                        T *const ptr_array_parameters_received,
                                                                        T *const ptr_array_previous_biased_first_moment_received,
                                                                        T *const ptr_array_previous_biased_second_moment_received)
{
    size_t const tmp_thread_global_index(blockIdx.x * blockDim.x + threadIdx.x);
    
    T tmp_partial_derivative,
       tmp_biased_first_moment,
       tmp_biased_second_moment;

    if(tmp_thread_global_index < size_received)
    {
        tmp_partial_derivative = ptr_array_partial_derivative_received[tmp_thread_global_index];

        ptr_array_previous_biased_first_moment_received[tmp_thread_global_index] = tmp_biased_first_moment = beta1_received * ptr_array_previous_biased_first_moment_received[tmp_thread_global_index] + (T(1) - beta1_received) * tmp_partial_derivative;
        ptr_array_previous_biased_second_moment_received[tmp_thread_global_index] = tmp_biased_second_moment = beta2_received * ptr_array_previous_biased_second_moment_received[tmp_thread_global_index] + (T(1) - beta2_received) * tmp_partial_derivative * tmp_partial_derivative;

        ptr_array_parameters_received[tmp_thread_global_index] -= learning_rate_at_time_t_received * tmp_biased_first_moment / (sqrt(tmp_biased_second_moment) + epsilon_received) + ptr_array_connections_mask_rergularization_received[tmp_thread_global_index] * weight_decay_received * ptr_array_parameters_received[tmp_thread_global_index];

        ptr_array_partial_derivative_received[tmp_thread_global_index] = T(0);
    }
}

template<typename T>
__global__ void kernel_while__Update_Parameter__Adam(size_t const size_received,
                                                                                T const weight_decay_received,
                                                                                T const beta1_received,
                                                                                T const beta2_received,
                                                                                T const epsilon_received,
                                                                                T const learning_rate_at_time_t_received,
                                                                                T const *const ptr_array_connections_mask_rergularization_received,
                                                                                T *const ptr_array_partial_derivative_received,
                                                                                T *const ptr_array_parameters_received,
                                                                                T *const ptr_array_previous_biased_first_moment_received,
                                                                                T *const ptr_array_previous_biased_second_moment_received)
{
    size_t tmp_thread_global_index(blockIdx.x * blockDim.x + threadIdx.x);
    
    T tmp_partial_derivative,
       tmp_biased_first_moment,
       tmp_biased_second_moment;

    do
    {
        tmp_partial_derivative = ptr_array_partial_derivative_received[tmp_thread_global_index];

        ptr_array_previous_biased_first_moment_received[tmp_thread_global_index] = tmp_biased_first_moment = beta1_received * ptr_array_previous_biased_first_moment_received[tmp_thread_global_index] + (T(1) - beta1_received) * tmp_partial_derivative;
        ptr_array_previous_biased_second_moment_received[tmp_thread_global_index] = tmp_biased_second_moment = beta2_received * ptr_array_previous_biased_second_moment_received[tmp_thread_global_index] + (T(1) - beta2_received) * tmp_partial_derivative * tmp_partial_derivative;

        ptr_array_parameters_received[tmp_thread_global_index] -= learning_rate_at_time_t_received * tmp_biased_first_moment / (sqrt(tmp_biased_second_moment) + epsilon_received) + ptr_array_connections_mask_rergularization_received[tmp_thread_global_index] * weight_decay_received * ptr_array_parameters_received[tmp_thread_global_index];

        ptr_array_partial_derivative_received[tmp_thread_global_index] = T(0);

        tmp_thread_global_index += gridDim.x * blockDim.x;
    } while(tmp_thread_global_index < size_received);
}

__device__ void CUDA_Neural_Network::Update_Parameter__Adam(size_t const batch_size_received, size_t const training_size_received, size_t const start_index_received, size_t const end_index_received)
{
    size_t i;

    T_ const tmp_learning_rate_scale(this->use_Warm_Restarts ? this->Warm_Restarts_Decay() / this->adam_learning_rate : 1_T);

    ++this->optimizer_time_step;

    T_ const *const tmp_ptr_array_connections_mask_rergularization(this->ptr_array_mask_regularized_parameters),
                  tmp_learning_rate(tmp_learning_rate_scale * this->adam_learning_rate),
                  tmp_weight_decay(this->use_normalized_weight_decay ? this->Normalized_Weight_Decay(batch_size_received, training_size_received) : this->regularization__weight_decay),
                  tmp_beta1(this->adam_beta1),
                  tmp_beta2(this->adam_beta2),
                  tmp_epsilon(this->adam_epsilon),
                  tmp_adam_epochs(this->optimizer_time_step),
                  tmp_learning_rate_at_time_t(tmp_learning_rate * sqrt(1_T - pow(tmp_beta2, tmp_adam_epochs)) / (1_T - pow(tmp_beta1, tmp_adam_epochs)));
    T_ *const tmp_ptr_array_partial_derivative(this->ptr_array_derivatives_parameters),
         *const tmp_ptr_array_parameters(this->ptr_array_parameters),
         *const tmp_ptr_array_previous_biased_first_moment(this->ptr_array_previous_biased_first_moment),
         *const tmp_ptr_array_previous_biased_second_moment(this->ptr_array_previous_biased_second_moment),
         tmp_partial_derivative,
         tmp_biased_first_moment,
         tmp_biased_second_moment;
    
    if(tmp_weight_decay != 0_T)
    {
        if(USE_PARALLEL && end_index_received - start_index_received >= warpSize)
        {
            // KERNEL LAUNCH
            //    1: Launching do-while elements.
            if(this->ptr_array_dim3_grid[1u].x * this->ptr_array_dim3_block[1u].x < end_index_received - start_index_received)
            {
                kernel_while__Update_Parameter__Adam<T_> <<< this->ptr_array_dim3_grid[1u], this->ptr_array_dim3_block[1u] >>> (end_index_received - start_index_received,
                                                                                                                                                                                            tmp_weight_decay,
                                                                                                                                                                                            tmp_beta1,
                                                                                                                                                                                            tmp_beta2,
                                                                                                                                                                                            tmp_epsilon,
                                                                                                                                                                                            tmp_learning_rate_at_time_t,
                                                                                                                                                                                            tmp_ptr_array_connections_mask_rergularization + start_index_received,
                                                                                                                                                                                            tmp_ptr_array_partial_derivative + start_index_received,
                                                                                                                                                                                            tmp_ptr_array_parameters + start_index_received,
                                                                                                                                                                                            tmp_ptr_array_previous_biased_first_moment + start_index_received,
                                                                                                                                                                                            tmp_ptr_array_previous_biased_second_moment + start_index_received);
            }
            //    2: Launching size condition.
            else if(this->ptr_array_dim3_grid[1u].x * this->ptr_array_dim3_block[1u].x > end_index_received - start_index_received)
            {
                kernel__Update_Parameter__Adam<T_> <<< this->ptr_array_dim3_grid[1u], this->ptr_array_dim3_block[1u] >>> (end_index_received - start_index_received,
                                                                                                                                                                                    tmp_weight_decay,
                                                                                                                                                                                    tmp_beta1,
                                                                                                                                                                                    tmp_beta2,
                                                                                                                                                                                    tmp_epsilon,
                                                                                                                                                                                    tmp_learning_rate_at_time_t,
                                                                                                                                                                                    tmp_ptr_array_connections_mask_rergularization + start_index_received,
                                                                                                                                                                                    tmp_ptr_array_partial_derivative + start_index_received,
                                                                                                                                                                                    tmp_ptr_array_parameters + start_index_received,
                                                                                                                                                                                    tmp_ptr_array_previous_biased_first_moment + start_index_received,
                                                                                                                                                                                    tmp_ptr_array_previous_biased_second_moment + start_index_received);
            }
            //    3: Standard.
            else
            {
                kernel__Update_Parameter__Adam<T_> <<< this->ptr_array_dim3_grid[1u], this->ptr_array_dim3_block[1u] >>> (tmp_weight_decay,
                                                                                                                                                                                    tmp_beta1,
                                                                                                                                                                                    tmp_beta2,
                                                                                                                                                                                    tmp_epsilon,
                                                                                                                                                                                    tmp_learning_rate_at_time_t,
                                                                                                                                                                                    tmp_ptr_array_connections_mask_rergularization + start_index_received,
                                                                                                                                                                                    tmp_ptr_array_partial_derivative + start_index_received,
                                                                                                                                                                                    tmp_ptr_array_parameters + start_index_received,
                                                                                                                                                                                    tmp_ptr_array_previous_biased_first_moment + start_index_received,
                                                                                                                                                                                    tmp_ptr_array_previous_biased_second_moment + start_index_received);
            }
            // |END| KERNEL LAUNCH |END|

            CUDA__Check_Error();
        }
        else
        {
            for(i = start_index_received; i != end_index_received; ++i)
            {
                tmp_partial_derivative = tmp_ptr_array_partial_derivative[i];

                tmp_ptr_array_previous_biased_first_moment[i] = tmp_biased_first_moment = tmp_beta1 * tmp_ptr_array_previous_biased_first_moment[i] + (1_T - tmp_beta1) * tmp_partial_derivative;
                tmp_ptr_array_previous_biased_second_moment[i] = tmp_biased_second_moment = tmp_beta2 * tmp_ptr_array_previous_biased_second_moment[i] + (1_T - tmp_beta2) * tmp_partial_derivative * tmp_partial_derivative;

                tmp_ptr_array_parameters[i] -= tmp_learning_rate_at_time_t * tmp_biased_first_moment / (sqrt(tmp_biased_second_moment) + tmp_epsilon) + tmp_ptr_array_connections_mask_rergularization[i] * tmp_weight_decay * tmp_ptr_array_parameters[i];

                tmp_ptr_array_partial_derivative[i] = 0_T;
            }
        }
    }
    else
    {
        if(USE_PARALLEL && end_index_received - start_index_received >= warpSize)
        {
            // KERNEL LAUNCH
            //    1: Launching do-while elements.
            if(this->ptr_array_dim3_grid[1u].x * this->ptr_array_dim3_block[1u].x < end_index_received - start_index_received)
            {
                kernel_while__Update_Parameter__Adam<T_> <<< this->ptr_array_dim3_grid[1u], this->ptr_array_dim3_block[1u] >>> (end_index_received - start_index_received,
                                                                                                                                                                                            tmp_beta1,
                                                                                                                                                                                            tmp_beta2,
                                                                                                                                                                                            tmp_epsilon,
                                                                                                                                                                                            tmp_learning_rate_at_time_t,
                                                                                                                                                                                            tmp_ptr_array_partial_derivative + start_index_received,
                                                                                                                                                                                            tmp_ptr_array_parameters + start_index_received,
                                                                                                                                                                                            tmp_ptr_array_previous_biased_first_moment + start_index_received,
                                                                                                                                                                                            tmp_ptr_array_previous_biased_second_moment + start_index_received);
            }
            //    2: Launching size condition.
            else if(this->ptr_array_dim3_grid[1u].x * this->ptr_array_dim3_block[1u].x > end_index_received - start_index_received)
            {
                kernel__Update_Parameter__Adam<T_> <<< this->ptr_array_dim3_grid[1u], this->ptr_array_dim3_block[1u] >>> (end_index_received - start_index_received,
                                                                                                                                                                                    tmp_beta1,
                                                                                                                                                                                    tmp_beta2,
                                                                                                                                                                                    tmp_epsilon,
                                                                                                                                                                                    tmp_learning_rate_at_time_t,
                                                                                                                                                                                    tmp_ptr_array_partial_derivative + start_index_received,
                                                                                                                                                                                    tmp_ptr_array_parameters + start_index_received,
                                                                                                                                                                                    tmp_ptr_array_previous_biased_first_moment + start_index_received,
                                                                                                                                                                                    tmp_ptr_array_previous_biased_second_moment + start_index_received);
            }
            //    3: Standard.
            else
            {
                kernel__Update_Parameter__Adam<T_> <<< this->ptr_array_dim3_grid[1u], this->ptr_array_dim3_block[1u] >>> (tmp_beta1,
                                                                                                                                                                                    tmp_beta2,
                                                                                                                                                                                    tmp_epsilon,
                                                                                                                                                                                    tmp_learning_rate_at_time_t,
                                                                                                                                                                                    tmp_ptr_array_partial_derivative + start_index_received,
                                                                                                                                                                                    tmp_ptr_array_parameters + start_index_received,
                                                                                                                                                                                    tmp_ptr_array_previous_biased_first_moment + start_index_received,
                                                                                                                                                                                    tmp_ptr_array_previous_biased_second_moment + start_index_received);
            }
            // |END| KERNEL LAUNCH |END|

            CUDA__Check_Error();
        }
        else
        {
            for(i = start_index_received; i != end_index_received; ++i)
            {
                tmp_partial_derivative = tmp_ptr_array_partial_derivative[i];

                tmp_ptr_array_previous_biased_first_moment[i] = tmp_biased_first_moment = tmp_beta1 * tmp_ptr_array_previous_biased_first_moment[i] + (1_T - tmp_beta1) * tmp_partial_derivative;
                tmp_ptr_array_previous_biased_second_moment[i] = tmp_biased_second_moment = tmp_beta2 * tmp_ptr_array_previous_biased_second_moment[i] + (1_T - tmp_beta2) * tmp_partial_derivative * tmp_partial_derivative;

                tmp_ptr_array_parameters[i] -= tmp_learning_rate_at_time_t * tmp_biased_first_moment / (sqrt(tmp_biased_second_moment) + tmp_epsilon);

                tmp_ptr_array_partial_derivative[i] = 0_T;
            }
        }
    }
}
